#include "hip/hip_runtime.h"
#include "lci.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <string.h>
#include <unistd.h>

#include "comm_exp.h"

#undef MAX_MSG
#define MAX_MSG (8 * 1024)

int total = TOTAL;
int skip = SKIP;

__global__ void kernel(void* dst, void* src, unsigned int count)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned char* d = (unsigned char*)dst;
  unsigned char* s = (unsigned char*)src;
  if (i < count)
      d[i] = s[i];
}

int main(int argc, char** args) {
  LCI_initialize(&argc, &args);
  LCI_endpoint_t ep;
  LCI_PL_t prop;
  LCI_PL_create(&prop);
  LCI_MT_t mt;
  LCI_MT_create(0, &mt);
  LCI_PL_set_mt(&mt, &prop);
  LCI_endpoint_create(0, prop, &ep);

  int rank = LCI_RANK;
  int tag = 99;

  LCI_syncl_t sync;

  double t1 = 0;
  size_t alignment = sysconf(_SC_PAGESIZE);
  void* src_buf = 0;
  void* dst_buf = 0;
  void* dst_gpu_buf = 0;
  hipMalloc(&src_buf, MAX_MSG);
  posix_memalign(&dst_buf, alignment, MAX_MSG);
  hipMalloc(&dst_gpu_buf, MAX_MSG);

  if (rank == 0) {
    for (int size = MIN_MSG; size <= MAX_MSG; size <<= 1) {
      hipMemset(src_buf, 'a', size);
      memset(dst_buf, 'b', size);

      if (size > LARGE) { total = TOTAL_LARGE; skip = SKIP_LARGE; }

      for (int i = 0; i < total + skip; i++) {
        assert(hipGetLastError() == hipSuccess);
        if (i == skip) t1 = wtime();
        while (LCI_sendbc(src_buf, size, 1-rank, tag, ep) != LCI_OK)
          LCI_progress(0, 1);

        LCI_one2one_set_empty(&sync);
        LCI_recvbc(dst_buf, size, 1-rank, tag, ep, &sync);
        while (LCI_one2one_test_empty(&sync))
          LCI_progress(0, 1);

        hipMemcpy(dst_gpu_buf, dst_buf, size, hipMemcpyHostToDevice);
        kernel<<<max(size/64, 1), 64>>>(src_buf, dst_gpu_buf, size);
        hipDeviceSynchronize();
      }

      t1 = 1e6 * (wtime() - t1) / total / 2;
      printf("%10.d %10.3f\n", size, t1);
    }
  } else {
    for (int size = MIN_MSG; size <= MAX_MSG; size <<= 1) {
      hipMemset(src_buf, 'a', size);
      memset(dst_buf, 'b', size);

      if (size > LARGE) { total = TOTAL_LARGE; skip = SKIP_LARGE; }

      for (int i = 0; i < total + skip; i++) {
        assert(hipGetLastError() == hipSuccess);
        LCI_one2one_set_empty(&sync);
        LCI_recvbc(dst_buf, size, 1-rank, tag, ep, &sync);
        while (LCI_one2one_test_empty(&sync))
          LCI_progress(0, 1);

        hipMemcpy(dst_gpu_buf, dst_buf, size, hipMemcpyHostToDevice);
        kernel<<<max(size/64, 1), 64>>>(src_buf, dst_gpu_buf, size);
        hipDeviceSynchronize();

        while (LCI_sendbc(src_buf, size, 1-rank, tag, ep) != LCI_OK)
          LCI_progress(0, 1);
      }
    }
  }
  hipFree(src_buf);
  free(dst_buf);
  hipFree(dst_gpu_buf);
  LCI_finalize();
}
